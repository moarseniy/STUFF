#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../cu/kernels.h"


__global__
void AddVector() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
}

void StartAddVector() {
  AddVector<<<>>>
}
